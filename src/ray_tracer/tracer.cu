#include "hip/hip_runtime.h"
#include "tracer.h"

// little bit of GPU testing biatch

namespace CudaTracing {
    __global__ void test_kernel() {
        printf("Hello from the GPU\n");
    }

    __global__ void update(std::vector<Eigen::Vector3f> *rays, int N, Pose *start_pose, loop_closure::LoopClosureConfig *config) {
        
        //printf("Inside the kernel");
        // first: calculate the index and stride size for this operation
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for(int i = index; i < N; i+= stride) {
            
        }
    }

    int helloWorld() {
        test_kernel<<<1,1>>>();
        hipDeviceSynchronize();
        return 0;
    }

    /*
    * Function, which takes in the rays, calls upon the kernel function until ready
    */
    void updateRays(std::vector<Eigen::Vector3f> *rays, Pose *start_pose, loop_closure::LoopClosureConfig *config) {
        //printf("Inside the function running the kernel");
        // Run kernel on 1M elements on the GPU
        int blockSize = 256;
        int numBlocks = (rays->size() + blockSize - 1) / blockSize;
        update<<<numBlocks, blockSize>>>(rays, rays->size(), start_pose, config);
    }
}

// Kernel function to add the elements of two arrays
/*__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}*/



/*int main(void)
{
    int N = 1 << 20;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    std::cout << "num blocks: " << numBlocks << " Bloc-size: " << blockSize << std::endl;
    add<<<numBlocks, blockSize>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
    
    return 0;
}*/